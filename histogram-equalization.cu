#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define GPU_CUDA_CODE
#include "hist-equ.h"
#define BIN_SIZE 256



__global__ void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= img_size)
		return;

	int pixel = img_in[i]; //prefetching

	__shared__ int block_hist[256];
	if(threadIdx.x < 256)
		block_hist[threadIdx.x] = 0;
	__syncthreads();

	//count at each block
	atomicAdd(&block_hist[pixel],1);
	__syncthreads();

	//Total Sum
	if(threadIdx.x <256)
	 	atomicAdd( &hist_out[threadIdx.x], block_hist[threadIdx.x]);

	return;
}


__global__ void histogram_prefixsum(int * hist_in,int * cdf,int nbr_bin,int img_size/*,int TypeofAlgo*/){
	

	__shared__ int partialScan[BIN_SIZE];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	//init
	if(i < nbr_bin)
		partialScan[threadIdx.x] = hist_in[threadIdx.x];
	//partial scan implementation
	for(int stride = 1; stride < blockDim.x; stride <<= 1){
		__syncthreads();
		if(threadIdx.x >= stride){
			partialScan[threadIdx.x] += partialScan[threadIdx.x - stride];
		}
	}
	__syncthreads();

	cdf[i] = partialScan[i];
	

	//NOT EFFECTIVE CODE ALTHOUGH IT FUNCTIONS CORRECTLY
	// if (TypeofAlgo == 1){
	// 	__shared__ int partialScan[2 * BIN_SIZE];
	// 	int i = threadIdx.x + blockDim.x * blockIdx.x;

	// 	if (i < nbr_bin)
	// 		partialScan[threadIdx.x] = hist_in[threadIdx.x];
		
	// 	__syncthreads();
	// 	//Redution
	// 	for(int stride = 1; stride <= blockDim.x; stride <<= 1){
	// 		__syncthreads();
	// 		int index = (threadIdx.x + 1)*2*stride - 1;

	// 		if (index < blockDim.x)
	// 			partialScan[index] += partialScan[index - stride];
	// 	}

	// 	//Post-Reduction
	// 	for(int stride = BIN_SIZE / 4; stride > 0; stride >>= 1){
	// 		__syncthreads();
	// 		int index = (threadIdx.x + 1)*2*stride - 1;

	// 		if (index + stride < BIN_SIZE)
	// 			partialScan[index + stride] += partialScan[index];
	// 	}

	// 	__syncthreads();
	// 	cdf[i] = partialScan[i];
	// }
}

__global__ void histogram_calcdf(int * cdf,int * lut ,int img_size){
	//shared variables will get broadcasted
	__shared__ int min; 
	__shared__ int d; 
	min = cdf[0];	
	d = img_size - min;

	__shared__ int block_lut[BIN_SIZE];
	block_lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min)*255/d + 0.5);
    if(block_lut[threadIdx.x] < 0)
        block_lut[threadIdx.x] = 0;
    
	lut[threadIdx.x] = block_lut[threadIdx.x];
	return ;
}

__global__ void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
							int * lut, int img_size, int nbr_bin)
{	
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= img_size)
		return;
	//preferch and load to register
	int reg = lut[img_in[i]];
	if( reg > 255){
		img_out[i] = 255;
	}
	else{
		img_out[i] = (unsigned char)reg;
	}

	__syncthreads();
}
