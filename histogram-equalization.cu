#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define GPU_CUDA_CODE
#include "hist-equ.h"
#define BIN_SIZE 256


__global__ void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockIdx.x*gridDim.x;
	if(i >= img_size)
		return;
	// if(BLOCK_SIZE-BIN_SIZE >= threadIdx.x)
	// if(i < nbr_bin)
		// hist_out[i] = 0;
	__shared__ int block_hist[256];
	if(threadIdx.x < 256)
		block_hist[threadIdx.x] = 0;
	__syncthreads();

	//while(i < img_size){
		atomicAdd(&block_hist[img_in[i]],1);
	//	i += stride;
	//}
	__syncthreads();

	if(threadIdx.x <256)
	 	atomicAdd( &hist_out[threadIdx.x], block_hist[threadIdx.x]);
	// hist_out[threadIdx.x % 256] = 1;
	// hist_out[img_in[i]] ++;
	// hist_out + img_in[x]
	return;
}


//  __global__ void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
//  							int * hist_in, int img_size, int nbr_bin,int * cdf,int * lut)
// {
	
	
//  	 //int *lut; 
//  	 //hipMalloc(&lut,sizeof(int)*nbr_bin);            
//  	 //if(i == 0)
//  	 //	hist_in[20000] =400;
//  	 //cdf = 0;
//  	int i = threadIdx.x + blockIdx.x * blockDim.x;
//  	if(i >= img_size)
//  		return;

//  	int d=0;
// 	int min;
//  	//__shared__ int lut[BIN_SIZE];
//  	__shared__ int scanning[BIN_SIZE];

//  	/* Construct the LUT by calculating the CDF */

//  	//major unoptimised code
	
//  	/*if(threadIdx.x == 0){
//  		while(min == 0){
//  			min = hist_in[d++];
//  		}
		
//  	}
//  	__syncthreads();
//  	d = img_size - min;
// 	*/

//  	//Parallel scan 
//  	if(i < nbr_bin)
//  		scanning[threadIdx.x] = hist_in[threadIdx.x];
//  	unsigned int stride;
//  	for(stride = 1; stride < blockIdx.x; stride *= 2){
//  		__syncthreads();
 		
//  		if (stride <= threadIdx.x)
//  			scanning[threadIdx.x] += scanning[threadIdx.x - stride];
 		
//  	}

//  	__syncthreads(); 
// 	for(int k=0; k < nbr_bin; k++){
// 		cdf[k] = scanning[k];
// 	}
//  	/*if(stride <= threadIdx.x)
//  		cdf[threadIdx.x] += temp;
//  	}*/
	
	
// 		min = cdf[0];
// 		d = min - img_size;
		
// 		lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min)*255/d + 0.5);
// 		if(lut[threadIdx.x] < 0){
// 			lut[threadIdx.x] = 0;
// 		} 
	
//  	/* Get the result image */
//  	if(lut[img_in[i]] > 255){
//  		img_out[i] = 255;
//  	}
//  	else{
//  		img_out[i] = (unsigned char)lut[img_in[i]];
//  	}
// 	__syncthreads();
// }


// --------------------WORKING CODE---------------------//
 
__global__ void histogram_prefixsum(int * hist_in,int * cdf,int nbr_bin,int img_size){
	__shared__ int partialScan[BIN_SIZE];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	// if(i >= img_size)
	// 	return;
	//// Major unoptimised code	
	if(i < nbr_bin)
		partialScan[threadIdx.x] = hist_in[threadIdx.x];
	
	for(int stride = 1; stride < blockDim.x; stride *= 2){
		__syncthreads();

		if(threadIdx.x >= stride){
			partialScan[threadIdx.x] += partialScan[threadIdx.x - stride];
		}
	}
	__syncthreads();

	// for(int k=0; k<nbr_bin; k++){
		cdf[i] = partialScan[i];
	// }	

}

__global__ void histogram_calcdf(int * cdf,int * lut ,int img_size){
	int min = cdf[0];
	int d = img_size - min;
	lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min)*255/d + 0.5);
    if(lut[threadIdx.x] < 0)
        lut[threadIdx.x] = 0;
    
	return ;
}

__global__ void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
							int * lut, int img_size, int nbr_bin)
{	
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(lut[img_in[i]] > 255){
		img_out[i] = 255;
	}
	else{
		img_out[i] = (unsigned char)lut[img_in[i]];
	}

	__syncthreads();
}
