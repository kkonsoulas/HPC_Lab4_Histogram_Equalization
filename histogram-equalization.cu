#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define GPU_CUDA_CODE
#include "hist-equ.h"
#define BIN_SIZE 256



__global__ void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= img_size)
		return;

	int pixel = img_in[i]; //prefetching

	__shared__ int block_hist[256];
	if(threadIdx.x < 256)
		block_hist[threadIdx.x] = 0;
	__syncthreads();


	atomicAdd(&block_hist[pixel],1);

	__syncthreads();

	if(threadIdx.x <256)
	 	atomicAdd( &hist_out[threadIdx.x], block_hist[threadIdx.x]);

	return;
}


//  __global__ void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
//  							int * hist_in, int img_size, int nbr_bin,int * cdf,int * lut)
// {
	
	
//  	 //int *lut; 
//  	 //hipMalloc(&lut,sizeof(int)*nbr_bin);            
//  	 //if(i == 0)
//  	 //	hist_in[20000] =400;
//  	 //cdf = 0;
//  	int i = threadIdx.x + blockIdx.x * blockDim.x;
//  	if(i >= img_size)
//  		return;

//  	int d=0;
// 	int min;
//  	//__shared__ int lut[BIN_SIZE];
//  	__shared__ int scanning[BIN_SIZE];

//  	/* Construct the LUT by calculating the CDF */

//  	//major unoptimised code
	
//  	/*if(threadIdx.x == 0){
//  		while(min == 0){
//  			min = hist_in[d++];
//  		}
		
//  	}
//  	__syncthreads();
//  	d = img_size - min;
// 	*/

//  	//Parallel scan 
//  	if(i < nbr_bin)
//  		scanning[threadIdx.x] = hist_in[threadIdx.x];
//  	unsigned int stride;
//  	for(stride = 1; stride < blockIdx.x; stride *= 2){
//  		__syncthreads();
 		
//  		if (stride <= threadIdx.x)
//  			scanning[threadIdx.x] += scanning[threadIdx.x - stride];
 		
//  	}

//  	__syncthreads(); 
// 	for(int k=0; k < nbr_bin; k++){
// 		cdf[k] = scanning[k];
// 	}
//  	/*if(stride <= threadIdx.x)
//  		cdf[threadIdx.x] += temp;
//  	}*/
	
	
// 		min = cdf[0];
// 		d = min - img_size;
		
// 		lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min)*255/d + 0.5);
// 		if(lut[threadIdx.x] < 0){
// 			lut[threadIdx.x] = 0;
// 		} 
	
//  	/* Get the result image */
//  	if(lut[img_in[i]] > 255){
//  		img_out[i] = 255;
//  	}
//  	else{
//  		img_out[i] = (unsigned char)lut[img_in[i]];
//  	}
// 	__syncthreads();
// }


// --------------------WORKING CODE---------------------//
__global__ void histogram_prefixsum(int * hist_in,int * cdf,int nbr_bin,int img_size,int TypeofAlgo){
	
	if (TypeofAlgo == 0){
		__shared__ int partialScan[BIN_SIZE];
		int i = threadIdx.x + blockIdx.x * blockDim.x;

		//// Major unoptimised code	
		if(i < nbr_bin)
			partialScan[threadIdx.x] = hist_in[threadIdx.x];
		
		for(int stride = 1; stride < blockDim.x; stride <<= 1){
			__syncthreads();

			if(threadIdx.x >= stride){
				partialScan[threadIdx.x] += partialScan[threadIdx.x - stride];
			}
		}
		__syncthreads();

		cdf[i] = partialScan[i];
	}

	if (TypeofAlgo == 1){
		__shared__ int partialScan[2 * BIN_SIZE];
		int i = threadIdx.x + blockDim.x * blockIdx.x;

		if (i < nbr_bin)
			partialScan[threadIdx.x] = hist_in[threadIdx.x];
		
		__syncthreads();
		//Redution
		for(int stride = 1; stride <= blockDim.x; stride <<= 1){
			__syncthreads();
			int index = (threadIdx.x + 1)*2*stride - 1;

			if (index < blockDim.x)
				partialScan[index] += partialScan[index - stride];
		}

		//Post-Reduction
		for(int stride = BIN_SIZE / 4; stride > 0; stride >>= 1){
			__syncthreads();
			int index = (threadIdx.x + 1)*2*stride - 1;

			if (index + stride < BIN_SIZE)
				partialScan[index + stride] += partialScan[index];
		}

		__syncthreads();
		cdf[i] = partialScan[i];
	}
}

__global__ void histogram_calcdf(int * cdf,int * lut ,int img_size){
	__shared__ int min; 
	__shared__ int d; 
	min = cdf[0];
	d = img_size - min;

	__shared__ int block_lut[BIN_SIZE];
	block_lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min)*255/d + 0.5);
    if(block_lut[threadIdx.x] < 0)
        block_lut[threadIdx.x] = 0;
    
	lut[threadIdx.x] = block_lut[threadIdx.x];
	return ;
}

__global__ void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
							int * lut, int img_size, int nbr_bin)
{	
	// __constant__ int quick_lut[BIN_SIZE];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= img_size)
		return;
	int reg = lut[img_in[i]];
	if( reg > 255){
		img_out[i] = 255;
	}
	else{
		img_out[i] = (unsigned char)reg;
	}

	__syncthreads();
}
