#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define GPU_CUDA_CODE
#include "hist-equ.h"
#define BIN_SIZE 256


__global__ void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockIdx.x*gridDim.x;
	if(i >= img_size)
		return;
	// if(BLOCK_SIZE-BIN_SIZE >= threadIdx.x)
	// if(i < nbr_bin)
		// hist_out[i] = 0;
	__shared__ int block_hist[256];
	if(threadIdx.x < 256)
		block_hist[threadIdx.x] = 0;
	__syncthreads();

	//while(i < img_size){
		atomicAdd(&block_hist[img_in[i]],1);
	//	i += stride;
	//}
	__syncthreads();

	if(threadIdx.x <256)
	 	atomicAdd( &hist_out[threadIdx.x], block_hist[threadIdx.x]);
	// hist_out[threadIdx.x % 256] = 1;
	// hist_out[img_in[i]] ++;
	// hist_out + img_in[x]
	return;
}


__global__ void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
							int * hist_in, int img_size, int nbr_bin){
	
	
	// int *lut; 
	// hipMalloc(&lut,sizeof(int)*nbr_bin);            
	// if(i == 0)
	// 	hist_in[20000] =400;
	// cdf = 0;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= img_size)
		return;

	int d=0;
   	__shared__ int lut[BIN_SIZE];
	__shared__ int min;
	__shared__ int cdf[BIN_SIZE];

	/* Construct the LUT by calculating the CDF */
	// d = 0;
	min = 0;

	// //major unoptimised code
	
	if(threadIdx.x == 0){
		while(min == 0){
			min = hist_in[d++];
		}
		
	}
	__syncthreads();
	d = img_size - min;
	

	// Parallel scan 
	if(threadIdx.x < nbr_bin){
		cdf[threadIdx.x] = hist_in[threadIdx.x];
		int stride,temp;
		for(stride = 1; stride < blockIdx.x; stride *= 2){
			__syncthreads();
			temp=0;
			if (stride <= threadIdx.x)
				temp = cdf[threadIdx.x - stride];
			__syncthreads();
			cdf[threadIdx.x] += temp;
		}

		__syncthreads();
		if(stride <= threadIdx.x)
			cdf[threadIdx.x] += temp;

		__syncthreads();	
		lut[threadIdx.x] = (int)(((float) cdf[threadIdx.x] - min)*255/d + 0.5);
		if(lut[threadIdx.x] < 0){
			lut[threadIdx.x] = 0;
		}
	}
	__syncthreads();
	/* Get the result image */

	if(lut[img_in[i]] > 255){
		img_out[i] = 255;
	}
	else{
		img_out[i] = (unsigned char)lut[img_in[i]];
	}

}
