#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define GPU_CUDA_CODE
#include "hist-equ.h"

// __device__ __constant__ unsigned char image_chunck[CONSTANT_MEMORY_SIZE];



PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{

    //init gpu
    int* d_hist;
    hipError_t hipError_t = hipMalloc((void**) &d_hist,256 * sizeof(int));
    int* cdf;
    hipError_t = hipMalloc((void**) &cdf,256 * sizeof(int));
    int* lut;
    hipError_t = hipMalloc((void**) &lut,256 * sizeof(int));
    unsigned char* image_chunck;
    hipError_t = hipMalloc((void**) &image_chunck,CONSTANT_MEMORY_SIZE * sizeof(unsigned char));
    cudaCheckError();
    // __constant__ int *quick_lut;
    // hipError_t = hipMalloc((void**) &quick_lut,256 * sizeof(int));
    int img_size = img_in.w  *img_in.h;
    dim3 block_dim , grid_dim;
    int i;
    //image larger than a block
    if(img_size > BLOCK_SIZE){
        block_dim.x = BLOCK_SIZE;
        grid_dim.x = img_size / BLOCK_SIZE;
        grid_dim.x+= img_size % BLOCK_SIZE ? 1 : 0; //image does not fite perfect
    }
    else{//image smaller than a block
        block_dim.x = img_size;
        grid_dim.x = 1;
    }
    // block_dim.x = img_size > BLOCK_SIZE ? BLOCK_SIZE : img_size;
    // grid_dim.x = img_size > BLOCK_SIZE ? img_size / BLOCK_SIZE : 1;
    // grid_dim.x+= img_size % BLOCK_SIZE ?  : 0;
    // int remainingThreads = img_size > BLOCK_SIZE ? img_size % BLOCK_SIZE : 0;





    //init
    PGM_IMG result;
    result.w = img_in.w;
    result.h = img_in.h;
    hipHostMalloc(&(result.img),result.w * result.h * sizeof(unsigned char));
    cudaCheckError();
    

    //timer
    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    

    hipMemset(d_hist,0,sizeof(int)*256);
    cudaCheckError();


    //Tilling Execution
    // for(i = 0; i <((img_in.w  *img_in.h) / CONSTANT_MEMORY_SIZE) ;i++){
    //     hipMemcpy(image_chunck,&img_in.img[i*CONSTANT_MEMORY_SIZE],CONSTANT_MEMORY_SIZE*sizeof(unsigned char),hipMemcpyHostToDevice);
    //     // hipMemcpyToSymbol(HIP_SYMBOL(image_chunck),&img_in.img[i*CONSTANT_MEMORY_SIZE],CONSTANT_MEMORY_SIZE*sizeof(unsigned char));
    //     cudaCheckError();
    //     // printf("i: %d\n",i);
    //     hipDeviceSynchronize();
    //     histogram<<<grid_dim,BLOCK_SIZE>>>(d_hist, image_chunck, CONSTANT_MEMORY_SIZE, 256);
    //     img_size -= CONSTANT_MEMORY_SIZE;
    // }
    // if(img_size > 0){
    //     hipMemcpy(image_chunck,&img_in.img[i*CONSTANT_MEMORY_SIZE],img_size*sizeof(unsigned char),hipMemcpyHostToDevice);
    //     // hipMemcpyToSymbol(HIP_SYMBOL(image_chunck),&img_in.img[i*CONSTANT_MEMORY_SIZE],img_size*sizeof(unsigned char));
    //     int grid_size = (img_size / BLOCK_SIZE) + 1;
    //     histogram<<<grid_size,1024>>>(d_hist, image_chunck, img_size, 256);
    // }
    // img_size = img_in.w  *img_in.h;
    
    //Non tilling execution
    histogram<<<grid_dim,block_dim>>>(d_hist, img_in.img, img_in.h * img_in.w, 256);

    //----WORKING-CODE------//
    hipDeviceSynchronize();
    cudaCheckError();
    histogram_prefixsum<<< 1, 256>>>(d_hist,cdf,256,  img_in.h * img_in.w, 0);
    hipDeviceSynchronize();
    cudaCheckError();
    histogram_calcdf<<< 1 , 256 >>>(cdf,lut,img_in.h * img_in.w);
    cudaCheckError();
    //-----------------------//

    hipDeviceSynchronize();
    // hipMemcpy(quick_lut,lut,256*sizeof(int),hipMemcpyHostToDevice);
    // hipMemcpy(&minimum,cdf,sizeof(int),hipMemcpyDeviceToDevice);
    
	// hipMemcpyToSymbol(HIP_SYMBOL(quick_lut),lut,256*sizeof(int));
	// hipMemcpyToSymbol(HIP_SYMBOL(&minimum),cdf,sizeof(int),0,hipMemcpyDeviceToDevice);

    cudaCheckError();
    histogram_equalization<<<grid_dim,block_dim>>>(result.img ,img_in.img ,lut,result.w*result.h , 256);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    //timer
    cudaCheckError();
    printf("Elapsed time in GPU:  %3.12f ms \n", time);

    hipFree(lut);
    hipFree(cdf);
    hipFree(image_chunck);
    hipFree(d_hist);
    return result;
}
