#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define GPU_CUDA_CODE
#include "hist-equ.h"

// __device__ __constant__ unsigned char image_chunck[CONSTANT_MEMORY_SIZE];

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{

    //init gpu
    int* d_hist;
    hipError_t hipError_t = hipMalloc((void**) &d_hist,256 * sizeof(int));
    int* cdf;
    hipError_t = hipMalloc((void**) &cdf,256 * sizeof(int));
    int* lut;
    hipError_t = hipMalloc((void**) &lut,256 * sizeof(int));
    #ifdef _TILLING_
    unsigned char* image_chunck;
    hipError_t = hipMalloc((void**) &image_chunck,CONSTANT_MEMORY_SIZE * sizeof(unsigned char));
    int i;
    #endif
    cudaCheckError();


    int img_size = img_in.w  *img_in.h;
    dim3 block_dim , grid_dim;


    //image larger than a block
    if(img_size > BLOCK_SIZE){
        block_dim.x = BLOCK_SIZE;
        grid_dim.x = img_size / BLOCK_SIZE;
        grid_dim.x+= img_size % BLOCK_SIZE ? 1 : 0; //image does not fite perfect
    }
    else{//image smaller than a block
        block_dim.x = img_size;
        grid_dim.x = 1;
    }


    //init
    PGM_IMG result;
    result.w = img_in.w;
    result.h = img_in.h;
    hipHostMalloc(&(result.img),result.w * result.h * sizeof(unsigned char));
    cudaCheckError();
    

    //timer
    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    

    hipMemset(d_hist,0,sizeof(int)*256);
    cudaCheckError();


    //Calculation of Histogram with Tilling
    #ifdef _TILLING_
    for(i = 0; i <((img_in.w  *img_in.h) / CONSTANT_MEMORY_SIZE) ;i++){
        hipMemcpy(image_chunck,&img_in.img[i*CONSTANT_MEMORY_SIZE],CONSTANT_MEMORY_SIZE*sizeof(unsigned char),hipMemcpyHostToDevice);
        // hipMemcpyToSymbol(HIP_SYMBOL(image_chunck),&img_in.img[i*CONSTANT_MEMORY_SIZE],CONSTANT_MEMORY_SIZE*sizeof(unsigned char));
        cudaCheckError();
        // printf("i: %d\n",i);
        hipDeviceSynchronize();
        histogram<<<grid_dim,BLOCK_SIZE>>>(d_hist, image_chunck, CONSTANT_MEMORY_SIZE, 256);
        img_size -= CONSTANT_MEMORY_SIZE;
    }
    if(img_size > 0){
        hipMemcpy(image_chunck,&img_in.img[i*CONSTANT_MEMORY_SIZE],img_size*sizeof(unsigned char),hipMemcpyHostToDevice);
        // hipMemcpyToSymbol(HIP_SYMBOL(image_chunck),&img_in.img[i*CONSTANT_MEMORY_SIZE],img_size*sizeof(unsigned char));
        int grid_size = (img_size / BLOCK_SIZE) + 1;
        histogram<<<grid_size,1024>>>(d_hist, image_chunck, img_size, 256);
    }
    img_size = img_in.w  *img_in.h;
    #else
    //Non tilling execution
    histogram<<<grid_dim,block_dim>>>(d_hist, img_in.img, img_in.h * img_in.w, 256);
    #endif

    hipDeviceSynchronize();
    cudaCheckError();
    
    histogram_prefixsum<<< 1, 256>>>(d_hist,cdf,256,  img_in.h * img_in.w);
    hipDeviceSynchronize();
    cudaCheckError();
    
    histogram_calcdf<<< 1 , 256 >>>(cdf,lut,img_in.h * img_in.w);
    hipDeviceSynchronize();
    cudaCheckError();


    histogram_equalization<<<grid_dim,block_dim>>>(result.img ,img_in.img ,lut,result.w*result.h , 256);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    //timer end

    cudaCheckError();
    printf("Elapsed time in GPU:  %3.6f ms \n", time);

    #ifdef _TILLING_
    hipFree(image_chunck);
    #endif
    hipFree(lut);
    hipFree(cdf);
    hipFree(d_hist);
    return result;
}
