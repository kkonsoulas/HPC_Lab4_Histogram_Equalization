#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define GPU_CUDA_CODE
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{

    //init gpu
    int* d_hist;
    hipError_t hipError_t = hipMalloc((void**) &d_hist,256 * sizeof(int));
    cudaCheckError();
    dim3 block_dim , grid_dim;
    int img_size = img_in.w  *img_in.h;
    int* cdf;
    hipError_t = hipMalloc((void**) &cdf,256 * sizeof(int));
    int* lut;
    hipError_t = hipMalloc((void**) &lut,256 * sizeof(int));
    
    //image larger than a block
    if(img_size > BLOCK_SIZE){
        block_dim.x = BLOCK_SIZE;
        grid_dim.x = img_size / BLOCK_SIZE;
        grid_dim.x+= img_size % BLOCK_SIZE ? 1 : 0; //image does not fite perfect
    }
    else{//image smaller than a block
        block_dim.x = img_size;
        grid_dim.x = 1;
    }
    // block_dim.x = img_size > BLOCK_SIZE ? BLOCK_SIZE : img_size;
    // grid_dim.x = img_size > BLOCK_SIZE ? img_size / BLOCK_SIZE : 1;
    // grid_dim.x+= img_size % BLOCK_SIZE ?  : 0;
    // int remainingThreads = img_size > BLOCK_SIZE ? img_size % BLOCK_SIZE : 0;





    //init
    PGM_IMG result;
    result.w = img_in.w;
    result.h = img_in.h;
    hipHostMalloc(&(result.img),result.w * result.h * sizeof(unsigned char));
    cudaCheckError();

    //timer
    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    

    hipMemset(d_hist,0,sizeof(int)*256);
    cudaCheckError();
    histogram<<<grid_dim,block_dim>>>(d_hist, img_in.img, img_in.h * img_in.w, 256);
    
    //----WORKING-CODE------//
    hipDeviceSynchronize();
    cudaCheckError();
    histogram_prefixsum<<< 1, 256>>>(d_hist,cdf,256,  img_in.h * img_in.w);
    hipDeviceSynchronize();
    cudaCheckError();
    histogram_calcdf<<< 1 , 256 >>>(cdf,lut,img_in.h * img_in.w);
    cudaCheckError();
    //-----------------------//

    hipDeviceSynchronize();
    cudaCheckError();
    histogram_equalization<<<grid_dim,block_dim>>>(result.img ,img_in.img ,lut,result.w*result.h , 256);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    //timer
    cudaCheckError();

    printf("Elapsed time in GPU:  %3.12f ms \n", time);
    return result;
}
